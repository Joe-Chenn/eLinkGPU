
#include <hip/hip_runtime.h>

__device__ void ion_match(int *mz, int *prefix, int *ion_dic,
                          int *ion_dic_prefix, int ion_dic_num,
                          float *matched, int *result_prefix, int64_t result_num,
                          int *max_idx_mass, int idx) {
    int s_id_l = prefix[idx],
        s_id_r = prefix[idx + 1];
    // if (idx == 210)
    //     printf("s_id_l: %d, s_id_r: %d\n", s_id_l, s_id_r);
    for (int i = s_id_l; i < s_id_r; i++) {
        if (mz[i] < ion_dic_num) {
            int mz_start = floor(mz[i] - mz[i] * 0.00002);
            int mz_end = ceil(mz[i] + mz[i] * 0.00002);
            // if (mz_start < 500) {
            //     mz_start = 500;
            // }
            // if (mz_end > 600000) {
            //     mz_end = 600000;
            // }
            // if (idx == 210)
            //     printf("mz_start: %d, mz_end: %d\n", mz_start, mz_end);

            for (int k = mz_start; k <= mz_end; k++) {
                int ll = ion_dic_prefix[k], rr = ion_dic_prefix[k + 1];

                for (int j = ll; j < rr; j++) {
                    // if (ion_dic[j] >= max_idx_mass[idx]) {
                    //     continue;
                    // }
                    int64_t map_key = result_prefix[idx] + ion_dic[j];
                    if (map_key < result_prefix[idx + 1]) {
                        
                        matched[map_key] += 1.0;
                    } else {
                        // if (idx == 210)
                        //     // printf("map_key: %ld\n", map_key);
                        //     printf("[ERROR] map_key: [%d, %d, %ld] out of range!",
                        //         result_prefix[idx], ion_dic[j], map_key);
                        break;
                    }
                }
            }
        }
    }
}

__device__ void calc_percentage(float *matched, int *result_prefix,
                                int *max_idx_mass, int *pep_ion_num, float *percentage,
                                int idx, int charge) {
    int left = result_prefix[idx], right = result_prefix[idx + 1];
    // if (idx == 2) {
    //     printf("left: %d, right: %d\n", left, right);
    // }
    for (int64_t i = left; i < right; i++) {

        percentage[i] = matched[i] == 0.0 ? 10000.0 : matched[i] / (pep_ion_num[i - left] * charge);
        // if (idx == 210)
        //     printf("percentage: %f\n", percentage[i]);
    }
}

__device__ void calc_bm25_score(int *mz, int *prefix, float *intensity, int *ion_dic,
                                int *ion_dic_prefix, int ion_dic_num, int64_t result_num, int *result_prefix,
                                int *max_idx_mass, int *pep_ion_num, float *percentage,
                                float *bm25_score, int idx) {
    int s_id_l = prefix[idx],
        s_id_r = prefix[idx + 1];
    float one_mz_start_bias, one_mz_end_bias, max_bias, one_mz_bias_score;
    for (int i = s_id_l; i < s_id_r; i++) {
        if (mz[i] < ion_dic_num) {
            float inten_score = sin(intensity[i] * 1.57075);
            int mz_start = floor(mz[i] - mz[i] * 0.00002);
            int mz_end = ceil(mz[i] + mz[i] * 0.00002);
            // if (mz_start < 500) {
            //     mz_start = 500;
            // }
            // if (mz_end > 6000) {
            //     mz_end = 6000;
            // }
            
            one_mz_start_bias = abs(mz_start - mz[i]);
            one_mz_end_bias = abs(mz_end - mz[i]);
            max_bias = one_mz_start_bias > one_mz_end_bias ? one_mz_start_bias : one_mz_end_bias;
            if (max_bias < 1) {
                continue;
            }
            for (int k = mz_start; k <= mz_end; k++) {
                int ll = ion_dic_prefix[k], rr = ion_dic_prefix[k + 1];
                one_mz_bias_score = log(2.718281828459 - abs(k - mz[i]) / max_bias);
                for (int j = ll; j < rr; j++) {
                    if (ion_dic[j] >= max_idx_mass[idx]) {
                        // printf("[ERROR] ion_dic[j]: %d, max_idx_mass: %d\n", ion_dic[j], max_idx_mass[idx]);
                        continue;
                    }
                    int64_t map_key = result_prefix[idx] + ion_dic[j];
                    if (map_key < result_prefix[idx + 1]) {
                        bm25_score[map_key] += one_mz_bias_score * inten_score * (1 + 0.001) / (inten_score + 0.001 * (percentage[map_key] * (-25) + 1 - (-25)));

                    } else {
                        // printf("[ERROR] map_key: [%d, %d, %d, %d] out of range!\n",
                        //        result_prefix[idx], ion_dic[j], max_idx_mass[idx], idx);
                        break;
                    }
                }
            }
        }
    }
}

__device__ void get_max_score_index(float *bm25_score, int *result_prefix, int *max_score_index, int idx) {
    int left = result_prefix[idx], right = result_prefix[idx + 1];
    float max_score = -1.0;
    int max_idx = -1;
    for (int i = left; i < right; i++) {
        if (bm25_score[i] > max_score) {
            max_score = bm25_score[i];
            max_idx = i;
        }
    }
    if (max_idx != -1) {
        max_score_index[idx] = max_idx;
    }
}

__device__ void get_candidate_num(float *bm25_score, float *matched, int *result_prefix, int *max_score_index, int *candidate_num,
                                  float filter_matched_value,
                                  float *pep_mass_list, int pep_mass_list_len,
                                  float *precursor_mass_list, float linker_mass, int *mass_index, int idx) {
    int left = result_prefix[idx], right = result_prefix[idx + 1];
    // int index_left = candidate_prefix[idx], index_right = candidate_prefix[idx + 1];
    float pep_1_mass, pep_2_mass_left, pep_2_mass_right, score;
    int pep2_mass_left_index, pep2_mass_right_index;
    int cnt = 0;
    if (idx == 210){
        printf("left: %d, right: %d\n", left, right);
        printf("max_score: %f\n", bm25_score[max_score_index[idx]]);
    }
    for (int i = left; i < right; i++) {
        if (bm25_score[i] > bm25_score[max_score_index[idx]] / 2 && matched[i] > filter_matched_value) {
            // if (index_left >= index_right) {
            //     printf("[ERROR] index_left: %d, index_right: %d, idx: %d, i: %d\n", index_left, index_right, idx, i);
            // }
            if (idx == 210)
                printf("pep_1_mass: %f\n", pep_1_mass);
            score = bm25_score[i];
            if (idx == 210)
                printf("score: %f\n", score);
            pep_1_mass = pep_mass_list[i - left];

            pep_2_mass_left = precursor_mass_list[idx] - precursor_mass_list[idx] * 0.00002 - linker_mass - pep_1_mass;
            pep_2_mass_right = precursor_mass_list[idx] + precursor_mass_list[idx] * 0.00002 - linker_mass - pep_1_mass;
            if (idx == 210)
                printf("pep_2_mass_left: %f, pep_2_mass_right: %f\n", pep_2_mass_left, pep_2_mass_right);
            if (pep_2_mass_left > 6000.0 || pep_2_mass_right < 500.0) {
                // printf("[ERROR] pep_2_mass_left: %f, pep_2_mass_right: %f, idx: %d, i: %d\n", pep_2_mass_left, pep_2_mass_right, idx, i);
                continue;
            }
            if (pep_2_mass_left < 500.0) {
                pep_2_mass_left = 500.0;
            }
            if (pep_2_mass_right > 6000.0) {
                pep_2_mass_right = 6000.0;
            }
            pep2_mass_left_index = mass_index[(int)pep_2_mass_left - 1];
            pep2_mass_right_index = mass_index[(int)pep_2_mass_right];

            if (pep2_mass_left_index > pep2_mass_right_index) {
                printf("[ERROR] pep2_mass: %f, pep2_mass_left_index: %d, pep2_mass_right_index: %d, idx: %d, i: %d\n", pep_2_mass_left, pep2_mass_left_index, pep2_mass_right_index, idx, i);
            }

            while (1) {
                if (pep2_mass_left_index > pep_mass_list_len) {
                    printf("[ERROR] pep2_mass: %f, pep2_mass_left_index: %d, pep_mass_list_len: %d, idx: %d, i: %d\n", pep_2_mass_left, pep2_mass_left_index, pep_mass_list_len, idx, i);
                    break;
                }
                if (pep_mass_list[pep2_mass_left_index] > pep_2_mass_right) {
                    break;
                }
                if (left + pep2_mass_left_index >= right) {
                    printf("[ERROR] precursor_mass: %f, pep1_mass: %f, pep2_mass: %f, pep2_mass_left_index: %d, right: %d, idx: %d, i: %d\n", precursor_mass_list[idx], pep_1_mass, pep_2_mass_left, pep2_mass_left_index, right, idx, i);
                }
                if (pep_mass_list[pep2_mass_left_index] < pep_2_mass_left) {
                    pep2_mass_left_index++;
                    continue;
                }

                // if (idx == 210 && pep2_mass_left_index == 88184) {
                //     printf("score: %f, bm25_score: %f\n", score, bm25_score[left + pep2_mass_left_index]);
                // }
                if (score + bm25_score[left + pep2_mass_left_index] >= bm25_score[max_score_index[idx]]) {
                    if (idx == 210) {
                        printf("pep_2_mass_left: %d, pep_2_mass_right: %d\n", pep2_mass_left_index, pep2_mass_right_index);
                        printf("score: %f, max_score: %f\n", score + bm25_score[left + pep2_mass_left_index], bm25_score[max_score_index[idx]]);
                        // printf("1\n");
                    }
                    cnt++;
                }
                pep2_mass_left_index++;
            }
            // index_left++;
        }
        
    }
    // if (idx == 210){
    //     printf("cnt: %d\n", cnt);
    //     printf("left: %d, right: %d\n", left, right);
    // }
    candidate_num[idx] = cnt;
}

__global__ void compute_ion_match(int *no_linker_mz, int *no_linker_mz_prefix, int spectrum_num,
                                  int *linker_mz, int *linker_mz_prefix,
                                  float *no_linker_intensity, float *linker_intensity,
                                  int *ion_dic, int *ion_dic_prefix, int ion_dic_num,
                                  float *matched, int *result_prefix, int64_t result_num,
                                  int *max_idx_mass, int *pep_ion_num, float *percentage, float *bm25_score,
                                  int *charge, int *max_score_index, int *candidate_num, float filter_matched_value,
                                  float *pep_mass_list, int pep_mass_list_len,
                                  float *precursor_mass_list, float linker_mass, int *mass_index) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx + 1 < spectrum_num) {

        ion_match(no_linker_mz, no_linker_mz_prefix,
                  ion_dic, ion_dic_prefix, ion_dic_num, matched, result_prefix,
                  result_num, max_idx_mass, idx);
        ion_match(linker_mz, linker_mz_prefix,
                  ion_dic, ion_dic_prefix, ion_dic_num, matched, result_prefix,
                  result_num, max_idx_mass, idx);

        calc_percentage(matched, result_prefix, max_idx_mass, pep_ion_num, percentage, idx, charge[idx]);

        calc_bm25_score(no_linker_mz, no_linker_mz_prefix, no_linker_intensity,
                        ion_dic, ion_dic_prefix, ion_dic_num, result_num, result_prefix,
                        max_idx_mass, pep_ion_num, percentage, bm25_score, idx);
        calc_bm25_score(linker_mz, linker_mz_prefix, linker_intensity,
                        ion_dic, ion_dic_prefix, ion_dic_num, result_num, result_prefix,
                        max_idx_mass, pep_ion_num, percentage, bm25_score, idx);

        get_max_score_index(bm25_score, result_prefix, max_score_index, idx);
        get_candidate_num(bm25_score, matched, result_prefix, max_score_index, candidate_num,
                          filter_matched_value, pep_mass_list, pep_mass_list_len,
                          precursor_mass_list, linker_mass, mass_index, idx);
        // if (idx == 19999)
        //     printf("1111");
    }
}

__global__ void get_valid_candidate_num(float *bm25_score, short *matched, int *result_prefix,
                                        int spectrum_num, int *max_score_index, short filter_matched_value,
                                        int *return_num_list, float *pep_mass_list, int pep_mass_list_len,
                                        float *precursor_mass_list, float linker_mass, int *mass_index) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx + 1 < spectrum_num) {
        int left = result_prefix[idx], right = result_prefix[idx + 1];
        // int index_left = candidate_prefix[idx], index_right = candidate_prefix[idx + 1];
        float pep_1_mass, pep_2_mass_left, pep_2_mass_right, score;
        int pep2_mass_left_index, pep2_mass_right_index;
        int cnt = 0;
        for (int i = left; i < right; i++) {
            if (bm25_score[i] > bm25_score[max_score_index[idx]] / 2 && matched[i] > filter_matched_value) {
                // if (index_left >= index_right) {
                //     printf("[ERROR] index_left: %d, index_right: %d, idx: %d, i: %d\n", index_left, index_right, idx, i);
                // }

                score = bm25_score[i];
                pep_1_mass = pep_mass_list[i - left];
                pep_2_mass_left = precursor_mass_list[idx] - precursor_mass_list[idx] * 0.00002 - linker_mass - pep_1_mass;
                pep_2_mass_right = precursor_mass_list[idx] + precursor_mass_list[idx] * 0.00002 - linker_mass - pep_1_mass;
                if (pep_2_mass_left > 6000.0 || pep_2_mass_right < 500.0) {
                    printf("[ERROR] pep_2_mass_left: %f, pep_2_mass_right: %f, idx: %d, i: %d\n", pep_2_mass_left, pep_2_mass_right, idx, i);
                }
                if (pep_2_mass_left < 500.0) {
                    pep_2_mass_left = 500.0;
                }
                if (pep_2_mass_right > 6000.0) {
                    pep_2_mass_right = 6000.0;
                }
                pep2_mass_left_index = mass_index[(int)pep_2_mass_left - 1];
                pep2_mass_right_index = mass_index[(int)pep_2_mass_right];
                if (pep2_mass_left_index > pep2_mass_right_index) {
                    printf("[ERROR] pep2_mass: %f, pep2_mass_left_index: %d, pep2_mass_right_index: %d, idx: %d, i: %d\n", pep_2_mass_left, pep2_mass_left_index, pep2_mass_right_index, idx, i);
                }

                while (1) {
                    if (pep2_mass_left_index > pep_mass_list_len) {
                        printf("[ERROR] pep2_mass: %f, pep2_mass_left_index: %d, pep_mass_list_len: %d, idx: %d, i: %d\n", pep_2_mass_left, pep2_mass_left_index, pep_mass_list_len, idx, i);
                    }
                    if (pep_mass_list[pep2_mass_left_index] < pep_2_mass_left) {
                        pep2_mass_left_index++;
                        continue;
                    }
                    if (pep_mass_list[pep2_mass_left_index] > pep_2_mass_right) {
                        break;
                    }

                    
                    score += bm25_score[pep2_mass_left_index];
                    if (score >= bm25_score[max_score_index[idx]]) {
                        cnt++;
                    }
                }
                // index_left++;
            }
            return_num_list[idx] = cnt;
        }
    }
}

__global__ void get_candidate(float *bm25_score, float *matched, int *result_prefix,
                              int spectrum_num, int *max_score_index, float filter_matched_value,
                              float *candidate_score, int *candidate_1_index, int *candidate_2_index,
                              int *candidate_prefix,
                              float *pep_mass_list, int pep_mass_list_len, float *precursor_mass_list, float linker_mass,
                              int *mass_index) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx + 1 < spectrum_num) {
        // if (idx == 2000)
        //     printf("step1\n");
        
        int left = result_prefix[idx], right = result_prefix[idx + 1];
        int index_left = candidate_prefix[idx], index_right = candidate_prefix[idx + 1];
        int cnt = 0;
        float pep_1_mass, pep_2_mass_left, pep_2_mass_right, score;
        int pep2_mass_left_index, pep2_mass_right_index;
        // int cnt = 0;
        for (int i = left; i < right; i++) {
            if (bm25_score[i] > bm25_score[max_score_index[idx]] / 2 && matched[i] > filter_matched_value) {
                if (index_left >= index_right) {
                    break;
                    // printf("[ERROR] index_left: %d, index_right: %d\n", candidate_prefix[idx], candidate_prefix[idx + 1]);
                    // printf("[ERROR] index_left: %d, index_right: %d, idx: %d, i: %d\n", index_left, index_right, idx, i);
                }

                score = bm25_score[i];
                pep_1_mass = pep_mass_list[i - left];
                pep_2_mass_left = precursor_mass_list[idx] - precursor_mass_list[idx] * 0.00002 - linker_mass - pep_1_mass;
                pep_2_mass_right = precursor_mass_list[idx] + precursor_mass_list[idx] * 0.00002 - linker_mass - pep_1_mass;
                if (pep_2_mass_left > 6000.0 || pep_2_mass_right < 500.0) {
                    // printf("[ERROR] pep_2_mass_left: %f, pep_2_mass_right: %f, idx: %d, i: %d\n", pep_2_mass_left, pep_2_mass_right, idx, i);
                    continue;
                }
                if (pep_2_mass_left < 500.0) {
                    pep_2_mass_left = 500.0;
                }
                if (pep_2_mass_right > 6000.0) {
                    pep_2_mass_right = 6000.0;
                }
                pep2_mass_left_index = mass_index[(int)pep_2_mass_left - 1];
                pep2_mass_right_index = mass_index[(int)pep_2_mass_right];
                if (pep2_mass_left_index > pep2_mass_right_index) {
                    printf("[ERROR] pep2_mass_left_index: %d, pep2_mass_right_index: %d, idx: %d, i: %d\t", pep2_mass_left_index, pep2_mass_right_index, idx, i);
                    printf("pep2_mass_left: %f, pep2_mass_right: %f\n", pep_2_mass_left, pep_2_mass_right);
                }

                while (1) {
                    if (pep2_mass_left_index > pep_mass_list_len) {
                        printf("[ERROR] pep2_mass_left_index: %d, pep_mass_list_len: %d, idx: %d, i: %d\n", pep2_mass_left_index, pep_mass_list_len, idx, i);
                        break;
                    }
                    if (pep_mass_list[pep2_mass_left_index] > pep_2_mass_right) {
                        break;
                    }
                    if (left + pep2_mass_left_index >= right) {
                        printf("[ERROR] precursor_mass: %f, pep1_mass: %f, pep2_mass: %f, pep2_mass_left_index: %d, right: %d, idx: %d, i: %d\n", precursor_mass_list[idx], pep_1_mass, pep_2_mass_left, pep2_mass_left_index, right, idx, i);
                    }
                    if (pep_mass_list[pep2_mass_left_index] < pep_2_mass_left) {
                        pep2_mass_left_index++;
                        continue;
                    }

                    if (index_left >= index_right) {
                        break;
                        // printf("[ERROR] index_left: %d, index_right: %d, idx: %d, i: %d\n", index_left, index_right, idx, i);
                    }

                    // score += ;
                    if (score + bm25_score[left + pep2_mass_left_index] >= bm25_score[max_score_index[idx]]) {
                        // if (idx == 210) {
                        //     printf("pep_2_mass_left: %d, pep_2_mass_right: %d\n", pep2_mass_left_index, pep2_mass_right_index);
                        //     // printf("score: %f, max_score: %f\n", score + bm25_score[left + pep2_mass_left_index], bm25_score[max_score_index[idx]]);
                        // // printf("1\n");
                        // }
                        candidate_score[index_left] = score  + bm25_score[left + pep2_mass_left_index];
                        candidate_1_index[index_left] = i;
                        candidate_2_index[index_left] = pep2_mass_left_index;
                        index_left++;
                        cnt++;
                    }
                    pep2_mass_left_index++;
                }
                // index_left++;
            }
        }
        if (idx == 19202){
            printf("index_left: %d\n", index_left);
        }
    }
}